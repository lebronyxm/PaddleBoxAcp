#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

 Licensed under the Apache License, Version 2.0 (the "License");
 you may not use this file except in compliance with the License.
 You may obtain a copy of the License at

     http://www.apache.org/licenses/LICENSE-2.0

 Unless required by applicable law or agreed to in writing, software
 distributed under the License is distributed on an "AS IS" BASIS,
 WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 See the License for the specific language governing permissions and
 limitations under the License. */

#pragma once

#include <hip/hip_runtime.h>
#include <vector>
#include "paddle/fluid/operators/tdm_child_op.h"

namespace paddle {
namespace operators {

template <typename T, typename InfoT = int, typename OutT = int>
__global__ void Kernel_TDMChildInner(const size_t N, const T *input_data,
                                     const InfoT *tree_info_data,
                                     const int child_nums, const int length,
                                     OutT *child_data, OutT *leaf_mask_data) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  for (; idx < N; idx += blockDim.x * gridDim.x) {
    const int input_ids = idx / child_nums;
    const int child_ids = idx % child_nums;

    int start_tree_id = static_cast<int>(input_data[input_ids]) * length + 3;
    if ((input_data[input_ids] == 0 || tree_info_data[start_tree_id] == 0)) {
      child_data[idx] = 0;
      leaf_mask_data[idx] = 0;
    } else {
      OutT child_id =
          static_cast<OutT>(tree_info_data[start_tree_id + child_ids]);
      child_data[idx] = child_id;
      leaf_mask_data[idx] = static_cast<OutT>(
          tree_info_data[static_cast<int>(child_id) * length] == 0 ? 0 : 1);
    }
  }
}

template <typename T, typename InfoT = int, typename OutT = int>
void TDMChildInnerCUDA(const framework::ExecutionContext &context,
                       const LoDTensor &input, const LoDTensor &tree_info,
                       LoDTensor *child, LoDTensor *mask) {
  auto child_nums = context.Attr<int>("child_nums");
  auto info_dims = tree_info.dims();
  int node_nums = info_dims[0];
  int length = info_dims[1];

  int input_ids_num = input.numel();
  VLOG(4) << "TDM child op: input numel ->  " << input_ids_num;

  auto *input_data = input.data<T>();
  auto *tree_info_data = tree_info.data<InfoT>();

  auto *child_data = child->mutable_data<OutT>(context.GetPlace());
  auto *leaf_mask_data = mask->mutable_data<OutT>(context.GetPlace());

  auto stream = context.cuda_device_context().stream();

  size_t N = input_ids_num * child_nums;
  // kernel
  Kernel_TDMChildInner<T, InfoT, OutT><<<(N + 512 - 1) / 512, 512, 0, stream>>>(
      N, input_data, tree_info_data, child_nums, length, child_data,
      leaf_mask_data);
}

template <typename DeviceContext, typename T>
class TDMChildCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    auto *input_var = ctx.InputVar("X");
    auto *tree_info_var = ctx.InputVar("TreeInfo");

    auto &input_tensor = input_var->Get<LoDTensor>();
    const auto &input_type = input_tensor.type();
    bool input_type_match = input_type == framework::proto::VarType::INT32 ||
                            input_type == framework::proto::VarType::INT64;
    PADDLE_ENFORCE_EQ(input_type_match, true,
                      platform::errors::InvalidArgument(
                          "Input(X) holds the wrong type, it holds %s, but "
                          "desires to be %s or %s",
                          paddle::framework::DataTypeToString(input_type),
                          paddle::framework::DataTypeToString(
                              framework::proto::VarType::INT32),
                          paddle::framework::DataTypeToString(
                              framework::proto::VarType::INT64)));

    auto &tree_info_tensor = tree_info_var->Get<LoDTensor>();
    const auto &info_type = tree_info_tensor.type();
    bool info_type_match = info_type == framework::proto::VarType::INT32 ||
                           info_type == framework::proto::VarType::INT64;
    PADDLE_ENFORCE_EQ(
        info_type_match, true,
        platform::errors::InvalidArgument(
            "Input(TreeInfo) holds the wrong type, it holds %s, but "
            "desires to be %s or %s",
            paddle::framework::DataTypeToString(info_type),
            paddle::framework::DataTypeToString(
                framework::proto::VarType::INT32),
            paddle::framework::DataTypeToString(
                framework::proto::VarType::INT64)));

    auto *child_var = ctx.OutputVar("Child");
    auto *leaf_mask_var = ctx.OutputVar("LeafMask");
    auto *child_tensor = child_var->GetMutable<framework::LoDTensor>();
    auto *leaf_mask_tensor = leaf_mask_var->GetMutable<framework::LoDTensor>();

    auto output_type =
        static_cast<framework::proto::VarType::Type>(ctx.Attr<int>("dtype"));
    bool out_type_match = output_type == framework::proto::VarType::INT32 ||
                          output_type == framework::proto::VarType::INT64;
    PADDLE_ENFORCE_EQ(out_type_match, true,
                      platform::errors::InvalidArgument(
                          "Ouput(Child) & Output(LeafMask) holds the wrong "
                          "type, it holds %s, but "
                          "desires to be %s or %s",
                          paddle::framework::DataTypeToString(output_type),
                          paddle::framework::DataTypeToString(
                              framework::proto::VarType::INT32),
                          paddle::framework::DataTypeToString(
                              framework::proto::VarType::INT64)));

    if (info_type == framework::proto::VarType::INT32 &&
        output_type == framework::proto::VarType::INT32) {
      TDMChildInnerCUDA<T, int, int>(ctx, input_tensor, tree_info_tensor,
                                     child_tensor, leaf_mask_tensor);
    } else if (info_type == framework::proto::VarType::INT64 &&
               output_type == framework::proto::VarType::INT32) {
      TDMChildInnerCUDA<T, int64_t, int>(ctx, input_tensor, tree_info_tensor,
                                         child_tensor, leaf_mask_tensor);
    } else if (info_type == framework::proto::VarType::INT32 &&
               output_type == framework::proto::VarType::INT64) {
      TDMChildInnerCUDA<T, int, int64_t>(ctx, input_tensor, tree_info_tensor,
                                         child_tensor, leaf_mask_tensor);
    } else if (info_type == framework::proto::VarType::INT64 &&
               output_type == framework::proto::VarType::INT64) {
      TDMChildInnerCUDA<T, int64_t, int64_t>(
          ctx, input_tensor, tree_info_tensor, child_tensor, leaf_mask_tensor);
    }
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(
    tdm_child,
    paddle::operators::TDMChildCUDAKernel<paddle::platform::CUDADeviceContext,
                                          float>,
    paddle::operators::TDMChildCUDAKernel<paddle::platform::CUDADeviceContext,
                                          double>,
    paddle::operators::TDMChildCUDAKernel<paddle::platform::CUDADeviceContext,
                                          int>,
    paddle::operators::TDMChildCUDAKernel<paddle::platform::CUDADeviceContext,
                                          int64_t>);
